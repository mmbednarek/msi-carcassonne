#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/Decoder/Decoder.h>
#include <Carcassonne_GPU/RL/Network.h>
#include <Util_GPU/Time.h>
#include <fmt/core.h>
#include <spdlog/spdlog.h>

namespace carcassonne::rl {

template<typename T, typename... ARGS>
DEVHOST constexpr boost::shared_ptr<T> make_shared(ARGS... args) {
   return boost::shared_ptr<T>(new T(args...));
}

DEVHOST Network::Network(const caffe::NetParameter &net_param, const caffe::SolverParameter &solver_param)
 : m_net(make_shared<caffe::Net<float>>(net_param))
 , m_solver(solver_param)
 , m_input(m_net->blob_by_name("input_data"))
 , m_output(m_net->blob_by_name("output_probas"))
 , m_allowed_moves(g_board_width * g_board_height * 4 * 14)
//  , m_label(m_net->blob_by_name("label_value"))
{
   m_solver.net() = m_net;
}

DEVHOST game::FullMove Network::do_move(game::Game &g, TileType tile, float prob)
{
   static constexpr auto output_neuron_count =  g_board_width * g_board_height * 4 * 10;
   g.board_to_caffe_X(m_neuron_input);
   std::copy(m_neuron_input.begin(), m_neuron_input.end(), m_input->mutable_cpu_data());

   spdlog::debug("forward start");
   m_net->Forward();
   spdlog::debug("forward stop");

   my_std::span<float> out_span(m_output->mutable_cpu_data(), output_neuron_count);
   auto move = decoder::decode_move(g, tile, m_allowed_moves, out_span, prob);
   return move;
}

}// namespace carcassonne::rl