#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/AI/Node.h>
#include <Carcassonne_GPU/ScoreBoard.h>
#include <cmath>
#include <utility>
#include <vector>

namespace carcassonne::ai {

DEVHOST Node::Node(game::Game &&game, const Player &player, game::FullMove move)
    : m_game(std::move(game)), m_player(player), m_move(move), m_id(0) {}

DEVHOST Node::Node(NodeId id, game::Game &&game, const Player &player, game::FullMove move, NodeId parent_id)
    : m_game(std::move(game)), m_player(player), m_move(move), m_parent_id(parent_id), m_id(id) {}

DEVHOST double Node::UCT1(mb::size rollout_count) const noexcept {
   const auto player_wins = m_player_wins[static_cast<int>(m_player)];
   if (m_simulation_count == 0) {
      return 1.0;
   }
   return static_cast<double>(player_wins) / static_cast<double>(m_simulation_count) + g_C * std::sqrt(std::log(static_cast<double>(rollout_count)) / static_cast<double>(m_simulation_count));
}

DEVHOST void Node::add_child(NodeId id) noexcept {
   m_children.push_back(id);
}

DEVHOST Player Node::find_winner() {
   auto max_score_it = std::max_element(m_game.scores().begin(), m_game.scores().end(), [](PlayerScore lhs, PlayerScore rhs) {
      return lhs.score < rhs.score;
   });
   return max_score_it->player;
}

DEVHOST game::Game &Node::game() const noexcept {
   return m_game;
}

}// namespace carcassonne::ai