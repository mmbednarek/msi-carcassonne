#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/AI/DeepRLPlayer.h>
#include <Carcassonne_GPU/AI/Tree.h>
#define SPDLOG_FMT_EXTERNAL
#include <spdlog/spdlog.h>

namespace carcassonne::ai {

DEVHOST DeepRLPlayer::DeepRLPlayer(game::Game &game, Player player, carcassonne::rl::Network &net) : m_player(player),
                                                                                        m_tree(game, player),
                                                                                        m_player_count(game.player_count()),
                                                                                        m_network(net) {
   spdlog::info("deep rl: initialising agent");
   game.on_next_move([this](game::Game &game, Player player, game::FullMove last_move) {
      m_last_moves[static_cast<mb::size>(last_player(player, m_player_count))] = last_move;
      if (player != m_player)
         return;
      make_move(game);
   });
}

DEVHOST void DeepRLPlayer::prepare_tree(const game::Game &game) {
   auto player = m_player;
   NodeId node_id = 0;
   do {
      node_id = m_tree.find_node_by_move(node_id, m_last_moves[static_cast<mb::size>(player)]);
      if (node_id == 0)
         break;
      player = next_player(player, m_player_count);
   } while (player != m_player);

   if (node_id == 0) {
      spdlog::debug("deep rl: building MCTS tree from scratch");
      m_tree.reset(game, m_player);
      return;
   }

   spdlog::debug("deep rl: reusing existing MCTS tree node");
   m_tree.change_root(node_id);
}

DEVHOST void DeepRLPlayer::make_move(game::Game &game) noexcept {
   spdlog::info("deep rl: preparing move");

   prepare_tree(game);
   Context ctx{
           .tree = m_tree,
           .network = m_network,
   };
   run_mcts(ctx, 2000);
   auto tile = game.tile_set()[game.move_index()];
   auto best_move = choose_move(ctx, game.move_index(), m_player);
   m_last_moves[static_cast<int>(m_player)] = best_move;

   while (!game.can_place_tile_and_figure(best_move.x, best_move.y, best_move.rotation, tile, best_move.direction)) {
      spdlog::info("deep rl: selected incorrect move, running MCTS again");
      run_mcts(ctx, 1000);
      best_move = choose_move(ctx, game.move_index(), m_player);
      m_last_moves[static_cast<int>(m_player)] = best_move;
   }

   auto move = game.new_move(m_player);
   if (move.place_tile_at(best_move.x, best_move.y, best_move.rotation)) {
      // spdlog::error("deep rl: selected tile placement is not feasible: {}", res.msg());
      return;
   }

   if (move.phase() == game::MovePhase::Done)
      return;

   if (best_move.ignored_figure) {
      if (move.ignore_figure()) {
         // spdlog::error("deep rl: cannot ignore figure at this location: {}", res.msg());
      }
      return;
   }

   if (move.place_figure(best_move.direction)) {
      // spdlog::error("deep rl: error placing figure: {}", res.msg());
      return;
   }
}

}// namespace carcassonne::ai