#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/AI/MCTSPlayer.h>
#include <Carcassonne_GPU/AI/Tree.h>
#include <fmt/core.h>

namespace carcassonne::ai {

DEVHOST MCTSPlayer::MCTSPlayer(game::Game &game, Player player, SimulationType sim_type) : m_player(player), m_tree(game, player), m_player_count(game.player_count()), m_simulation_type(sim_type) {
   game.on_next_move([this](game::Game &game, Player player, game::FullMove last_move) {
      m_last_moves[static_cast<mb::size>(last_player(player, m_player_count))] = last_move;
      if (player != m_player)
         return;
      make_move(game);
   });
}

DEVHOST void MCTSPlayer::prepare_tree(const game::Game &game) {
   auto player = m_player;
   NodeId node_id = 0;
   do {
      node_id = m_tree.find_node_by_move(node_id, m_last_moves[static_cast<mb::size>(player)]);
      if (node_id == 0)
         break;
      player = next_player(player, m_player_count);
   } while(player != m_player);

   if (node_id == 0) {
      fmt::print("node not in MCTS building a new tree\n");
      m_tree.reset(game, m_player);
      return;
   }

   fmt::print("node found in existing MCTS tree!\n");
   m_tree.change_root(node_id);
}

DEVHOST void MCTSPlayer::make_move(game::Game &game) noexcept {
   prepare_tree(game);
   run_mcts(m_tree, 2000, m_simulation_type);
   auto best_move = choose_move(m_tree, game.move_index(), m_player);
   m_last_moves[static_cast<int>(m_player)] = best_move;

   auto move = game.new_move(m_player);
   if (move.place_tile_at(best_move.x, best_move.y, best_move.rotation)) {
      // fmt::print("[MCTS internal error] selected tile placement is not feasible: {}\n", res.msg());
      return;
   }

   if (move.phase() == game::MovePhase::Done)
      return;

   if (best_move.ignored_figure) {
      if (move.ignore_figure()) {
         // fmt::print("[MCTS internal error] cannot ignore figure at this point: {}\n", res.msg());
      }
      return;
   }

   if (move.place_figure(best_move.direction)) {
      // fmt::print("[MCTS internal error] error placing figure: {}\n", res.msg());
      return;
   }
}

}// namespace carcassonne::ai