#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/AI/Tree.h>
#include <cmath>
#include <fmt/core.h>
#include <queue>
#include <cassert>

namespace carcassonne::ai {

DEVHOST Tree::Tree(const game::Game &game, const Player &player) {
   m_nodes.emplace_back(game::Game{game}, player, game::FullMove{});
}

DEVHOST struct NodePair {
   NodeId parent_id;
   NodeId old_id;

   constexpr bool operator<(const NodePair &other) const {
      return old_id > other.old_id;
   }
};

DEVHOST void Tree::change_root(NodeId new_root_id) {
   std::priority_queue<NodePair> queue;
   queue.push(NodePair{0, new_root_id});

   mb::size node_count = 0;
   while (!queue.empty()) {
      auto pair = queue.top();
      queue.pop();

      assert(pair.old_id >= node_count);

      auto &node = m_nodes[pair.old_id];
      const auto &children = node.children();
      for (const auto child_id : children) {
         queue.push(NodePair{node_count, child_id});
      }
      node.clear_children();

      node.update_id(node_count);
      node.update_parent_id(pair.parent_id);

      if (pair.old_id != 0) {
         auto &parent_node = m_nodes[pair.parent_id];
         parent_node.add_child(node_count);
      }
      m_nodes[node_count] = std::move(node);
      ++node_count;
   }

   m_nodes.erase(m_nodes.begin() + static_cast<long>(node_count), m_nodes.end());
}

DEVHOST mb::size Tree::add_node(std::unique_ptr<game::Game> &&game, Player player, game::FullMove move, NodeId parent_id) {
   auto node_id = m_nodes.size();
   m_nodes.emplace_back(node_id, std::move(game), player, move, parent_id);
   m_nodes[parent_id].add_child(node_id);
   return node_id;
}

DEVHOST Node &Tree::node_at(NodeId id) noexcept {
   return m_nodes[id];
}

DEVHOST NodeId Tree::find_node_by_move(NodeId base_id, const game::FullMove &move) {
   auto &children = m_nodes[base_id].children();
   auto node_it = std::find_if(children.begin(), children.end(), [move, this](NodeId &node_id) {
      return m_nodes[node_id].move() == move;
   });
   if (node_it == children.end()) {
      return 0;
   }
   return *node_it;
}

DEVHOST void Tree::reset(const game::Game &game, Player player) {
   m_nodes.clear();
   m_nodes.emplace_back(game::Game{game}, player, game::FullMove{});
}


}// namespace carcassonne::ai