#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif

#include <Carcassonne_GPU/AI/HeuristicPlayer.h>
#include <Carcassonne_GPU/AI/RandomPlayer.h>
#include <Carcassonne_GPU/AI/MCTS.h>
#include <Carcassonne_GPU/AI/Tree.h>
#include <cassert>
#include <chrono>
#include <fmt/core.h>
#include <Util_GPU/Time.h>
#include <random>

namespace carcassonne::ai {

DEVHOST std::array<HeuristicPlayer, 4> g_heuristic_players{
        HeuristicPlayer(Player::Black),
        HeuristicPlayer(Player::Blue),
        HeuristicPlayer(Player::Red),
        HeuristicPlayer(Player::Yellow),
};

DEVHOST std::random_device g_rand_device;
DEVHOST std::mt19937 g_random_gen(g_rand_device());

DEVHOST std::array<RandomPlayer<>, 4> g_random_players{
        RandomPlayer(g_random_gen, Player::Black),
        RandomPlayer(g_random_gen, Player::Blue),
        RandomPlayer(g_random_gen, Player::Red),
        RandomPlayer(g_random_gen, Player::Yellow),
};

DEVHOST DEVHOST void simulate_random(Tree &tree, NodeId node_id) {
   auto parent_id = node_id;
   game::Game simulated_game{tree.node_at(node_id).game()};
   for (auto move_index = simulated_game->move_index(); move_index < g_max_moves; ++move_index) {
      auto current_player = simulated_game->current_player();
      auto full_move = g_random_players[static_cast<mb::size>(current_player)].make_move(*simulated_game);
      simulated_game->update(0);
      parent_id = tree.add_node(simulated_game->clone(), current_player, full_move, parent_id);
   }

   auto &leaf_node = tree.node_at(parent_id);
   auto winner = leaf_node.find_winner();
   leaf_node.mark_as_expanded();
   backpropagate(tree, parent_id, winner);
}

DEVHOST void simulate(Tree &tree, NodeId node_id) {
   auto parent_id = node_id;
   game::Game simulated_game{tree.node_at(node_id).game()};
   for (auto move_index = simulated_game->move_index(); move_index < g_max_moves; ++move_index) {
      auto current_player = simulated_game->current_player();
      auto full_move = g_heuristic_players[static_cast<mb::size>(current_player)].make_move(*simulated_game);
      simulated_game->update(0);

      parent_id = tree.add_node(simulated_game->clone(), current_player, full_move, parent_id);
   }

   auto &leaf_node = tree.node_at(parent_id);
   auto winner = leaf_node.find_winner();
   leaf_node.mark_as_expanded();
   backpropagate(tree, parent_id, winner);
}

DEVHOST void backpropagate(Tree &tree, NodeId node_id, Player winner) {
   while (node_id != g_root_node) {
      auto &node = tree.node_at(node_id);
      node.propagate(winner);
      node_id = node.parent_id();
   }
   tree.node_at(g_root_node).propagate(winner);
}

DEVHOST void expand(Tree &tree, NodeId node_id, SimulationType simulation_type) {
   if (tree.node_at(node_id).simulation_count() == 0) {
      switch (simulation_type) {
      case SimulationType::Heuristic: {
         auto start = util_GPU::unix_time();
         simulate(tree, node_id);
         fmt::print("simulation lasted {}ms\n", (util_GPU::unix_time() - start));
         break;
      }
      case SimulationType::Random:
         simulate_random(tree, node_id);
         break;
      }
   }

   auto &node_children = tree.node_at(node_id).children();
   if (node_children.empty())
      return;

   auto child_node = *node_children.begin();
   auto simulation_move = tree.node_at(child_node).move();

   auto &game = tree.node_at(node_id).game();
   const auto current_player = game.current_player();
   for (auto tile_location : game.moves()) {
      bool simulated_tile = false;
      if (tile_location.x == simulation_move.x && tile_location.y == simulation_move.y && tile_location.rotation == simulation_move.rotation) [[unlikely]] {
         simulated_tile = true;
      }

      auto game_clone = game.clone();
      auto move = game_clone->new_move(current_player);
      move->place_tile(tile_location);

      for (Direction figure_move : game_clone->figure_placements(tile_location.x, tile_location.y)) {
         if (simulated_tile && figure_move == simulation_move.direction && !simulation_move.ignored_figure) [[unlikely]]
            continue;

         auto game_clone_clone = game_clone->clone();

         {
            auto move_clone = move->clone(*game_clone_clone);
            move_clone->place_figure(figure_move);
         }
         game_clone_clone->update(0);

         const auto full_move = game::FullMove{
                 .x = tile_location.x,
                 .y = tile_location.y,
                 .rotation = tile_location.rotation,
                 .ignored_figure = false,
                 .direction = figure_move,
         };
         tree.add_node(std::move(game_clone_clone), current_player, full_move, node_id);
      }

      if (simulated_tile && simulation_move.ignored_figure)
         continue;

      move->ignore_figure();
      game_clone->update(0);
      {
         const auto full_move = game::FullMove{
                 .x = tile_location.x,
                 .y = tile_location.y,
                 .rotation = tile_location.rotation,
                 .ignored_figure = true,
         };
         tree.add_node(std::move(game_clone), current_player, full_move, node_id);
      }
   }

   tree.node_at(node_id).mark_as_expanded();
}

DEVHOST void run_selection(Tree &tree, SimulationType sim_type) {
   const auto rollout_count = tree.node_at(g_root_node).simulation_count();

   auto current_node_id = g_root_node;
   for (;;) {
      auto &current_node = tree.node_at(current_node_id);
      const auto &children = current_node.children();

      auto selected_child_it = std::max_element(
              children.begin(),
              children.end(),
              [&tree, rollout_count](NodeId lhs, NodeId rhs) -> bool {
                 return tree.node_at(lhs).UCT1(rollout_count) < tree.node_at(rhs).UCT1(rollout_count);
              });
      assert(selected_child_it != children.end());

      const auto child_id = *selected_child_it;
      auto &child_node = tree.node_at(child_id);

      if (!child_node.expanded()) {
         expand(tree, child_id, sim_type);
         return;
      }

      if (child_node.children().empty()) {
         backpropagate(tree, child_id, child_node.find_winning_player());
         return;
      }

      current_node_id = child_id;
   }
}

DEVHOST void run_mcts(Tree &tree, mb::i64 time_limit, SimulationType sim_type) {
   if (!tree.node_at(g_root_node).expanded()) {
      expand(tree, g_root_node, sim_type);
   }

   auto until = std::chrono::steady_clock::now() + std::chrono::milliseconds{time_limit};
   while (std::chrono::steady_clock::now() < until) {
      run_selection(tree, sim_type);
   }
}

DEVHOST game::FullMove choose_move(Tree &tree, int move_index, Player player) {
   auto &root_node = tree.node_at(g_root_node);
   const auto &children = root_node.children();
   auto max_sim_count_it = std::max_element(
           children.begin(),
           children.end(),
           [&tree](NodeId lhs, NodeId rhs) -> bool {
             return tree.node_at(lhs).simulation_count() < tree.node_at(rhs).simulation_count();
           });
   auto max_sim_count = tree.node_at(*max_sim_count_it).simulation_count();

   auto selected = std::max_element(
           children.begin(), children.end(), [&tree, player, max_sim_count](NodeId lhs, NodeId rhs) {
             auto lhs_sc = tree.node_at(lhs).simulation_count();
             auto rhs_sc = tree.node_at(rhs).simulation_count();
             if (lhs_sc != max_sim_count && rhs_sc == max_sim_count)
                return true;
             if (rhs_sc != max_sim_count)
                return false;
             return tree.node_at(lhs).player_wins(player) > tree.node_at(rhs).player_wins(player);
           });

   assert(selected != children.end());
   auto &node = tree.node_at(*selected);
   return node.move();
}

}// namespace carcassonne::ai