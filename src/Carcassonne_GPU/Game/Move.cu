#ifdef __HIPCC__
#define DEVHOST __device__ __host__
#else
#define DEVHOST
#endif
#define EXIT_SUCCESS 0
#define EXIT_FAILURE 1
#include <Carcassonne_GPU/Game/Game.h>
#include <Carcassonne_GPU/Game/Move.h>
#include <memory>

namespace carcassonne::game {

DEVHOST Player Move::player() const noexcept {
   return m_player;
}

DEVHOST TileType Move::tile_type() const noexcept {
   return m_tile_type;
}

DEVHOST bool Move::place_tile_at(int x, int y, mb::u8 rotation) noexcept {
   if (m_phase != MovePhase::PlaceTile)
      return EXIT_FAILURE;

   if (m_game.board().can_place_at(x, y, m_tile_type, rotation)) {
      // fmt::print("OU OU... x={}, y={}, rot={}\n", x, y, rotation);
      return EXIT_FAILURE;
   }

   m_x = x;
   m_y = y;
   m_rotation = rotation;
   m_phase = MovePhase::PlaceFigure;

   m_game.apply_tile(x, y, m_tile_type, rotation);

   if (m_game.player_figure_count(m_player) == 0 || std::all_of(g_directions.cbegin(), g_directions.cend(), [this](Direction d) { return !is_free(d); })) {
      m_phase = MovePhase::Done;
      m_game.notify_tour_finished(FullMove{
              .x = m_x,
              .y = m_y,
              .rotation = m_rotation,
              .ignored_figure = true,
      });
   }
   return EXIT_SUCCESS;
}

DEVHOST bool Move::place_figure(Direction d) noexcept {
   if (m_phase != MovePhase::PlaceFigure) {
      return EXIT_FAILURE;
   }

   if (!is_free(d)) {
      return EXIT_FAILURE;
   }

   auto edge = make_edge(m_x, m_y, d);

   double px, py;
   std::tie(px, py) = direction_position(TilePosition{m_x, m_y}, d);
   m_game.add_figure(Figure{
           .player = m_player,
           .x = px,
           .y = py,
           .tile_x = m_x,
           .tile_y = m_y,
           .edge = edge,
           .dir = d,
   });

   m_game.mutable_groups().assign(make_edge(m_x, m_y, d), m_player);

   if (is_side_direction(d) && m_game.groups().is_completed(edge)) {
      m_game.on_structure_completed(m_game.groups().group_of(edge));
   } else if (d == Direction::Middle && m_game.is_monastery_completed(m_x, m_y)) {
      m_game.on_monastery_completed(m_x, m_y, m_player);
   }

   m_phase = MovePhase::Done;
   m_game.notify_tour_finished(FullMove{
           .x = m_x,
           .y = m_y,
           .rotation = m_rotation,
           .ignored_figure = false,
           .direction = d,
   });
   return EXIT_SUCCESS;
}

DEVHOST bool Move::ignore_figure() noexcept {
   if (m_phase != MovePhase::PlaceFigure)
      return EXIT_FAILURE;
   m_phase = MovePhase::Done;
   m_game.notify_tour_finished(FullMove{
           .x = m_x,
           .y = m_y,
           .rotation = m_rotation,
           .ignored_figure = true,
   });
   return EXIT_SUCCESS;
}

DEVHOST MovePhase Move::phase() const noexcept {
   return m_phase;
}

DEVHOST TilePosition Move::position() const noexcept {
   return TilePosition{m_x, m_y};
}

DEVHOST bool Move::is_free(Direction d) const noexcept {
   return m_game.can_place_figure(m_x, m_y, d);
}

DEVHOST ecuda::unique_ptr<Move> Move::clone(Game &game) const noexcept {
   return ecuda::unique_ptr<Move>(new Move{game, m_player, m_tile_type, m_x, m_y});
}

DEVHOST bool Move::place_tile(TileMove tile_location) noexcept {
   return place_tile_at(tile_location.x, tile_location.y, tile_location.rotation);
}

}
// namespace carcassonne::game
